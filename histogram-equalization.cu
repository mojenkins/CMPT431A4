#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"
// CUDA Runtime
#include <hip/hip_runtime.h>
// Utility and system includes
#include <hip/hip_runtime_api.h>
// helper for shared that are common to CUDA Samples
#include <helper_functions.h>

__global__ void histogram_work(int img_size, unsigned char* gpu_img_in, int * gpu_hist){

    if (blockIdx.x*blockDim.x + threadIdx.x < 256){
        gpu_hist[blockIdx.x*blockDim.x + threadIdx.x] = 0;
    }

    if (blockIdx.x*blockDim.x + threadIdx.x < img_size){
        atomicAdd(&gpu_hist[gpu_img_in[blockIdx.x*blockDim.x + threadIdx.x]], 1);
    }
}

void gpu_histogram(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin){
    // Set up pointers for gpu device memory
    unsigned char * gpu_img_in;
    int  * gpu_hist; 

    // Allocate memory for img_in, hist_in, lut, and img_out
    hipMalloc( (void**)&gpu_img_in, img_size * sizeof(unsigned char) );
    hipMalloc( (void**)&gpu_hist, (nbr_bin) * sizeof(int) );

    // Copy img_in to gpu
    hipMemcpy( gpu_img_in, img_in, img_size * sizeof(unsigned char), hipMemcpyHostToDevice );
    
    // call kernel
    histogram_work<<<img_size/512+1,512>>>(img_size, gpu_img_in, gpu_hist);

    // copy back (using hipMemcpy) gpu_img_out to img_out
    hipMemcpy( hist_out, gpu_hist, nbr_bin * sizeof(int), hipMemcpyDeviceToHost );

    // free gpu memory
    hipFree(gpu_img_in);
    hipFree(gpu_hist);
}

__global__ void histogram_equilization_work(int img_size, int* gpu_lut, unsigned char* gpu_img_in, unsigned char* gpu_img_out){
	if (blockIdx.x*blockDim.x + threadIdx.x < img_size){
    	gpu_img_out[blockIdx.x*blockDim.x + threadIdx.x] = (gpu_lut[gpu_img_in[blockIdx.x*blockDim.x + threadIdx.x]] > 255) ? 255 : gpu_lut[gpu_img_in[blockIdx.x*blockDim.x + threadIdx.x]];
	}
}


void gpu_histogram_equalization(unsigned char * img_out,
                                unsigned char * img_in,
                                int * hist_in,
                                int img_size,
                                int nbr_bin) {
    
    int *lut = (int *)malloc(sizeof(int)*nbr_bin);
    int i, min, d, cdf;
    
    /* Construct the LUT by calculating the CDF */
    cdf = 0;
    min = 0;
    i = 0;
    
    while(min == 0){
        min = hist_in[i++];
    }
    
    d = img_size - min;
    
    for(i = 0; i < nbr_bin; i ++){
        cdf += hist_in[i];
        //lut[i] = (cdf - min)*(nbr_bin - 1)/d;
        lut[i] = (int)(((float)cdf - min)*255/d + 0.5);
        
        if(lut[i] < 0){
            lut[i] = 0;
        }
    }
    
    // Set up pointers for gpu device memory
    unsigned char * gpu_img_in, * gpu_img_out;
    int  * gpu_lut; //, * gpu_cdf;
    
    // Allocate memory for img_in, hist_in, lut, and img_out
    hipMalloc( (void**)&gpu_img_in, img_size * sizeof(unsigned char) );
    hipMalloc( (void**)&gpu_img_out, img_size * sizeof(unsigned char) );
    hipMalloc( (void**)&gpu_lut, (nbr_bin) * sizeof(int) );
    
    // Copy img_in and cdf to gpu
    hipMemcpy( gpu_img_in, img_in, img_size * sizeof(unsigned char), hipMemcpyHostToDevice );
    hipMemcpy( gpu_lut, lut, (nbr_bin) * sizeof(int), hipMemcpyHostToDevice );
    free(lut);
    
    // call kernel
    histogram_work<<<img_size/512+1,512>>>(img_size, gpu_lut, gpu_img_in, gpu_img_out);
    
    // copy back (using hipMemcpy) gpu_img_out to img_out
    hipMemcpy( img_out, gpu_img_out, img_size * sizeof(unsigned char), hipMemcpyDeviceToHost );
    
    // free gpu memory
    hipFree(gpu_img_out);
    hipFree(gpu_img_in);
    hipFree(gpu_lut);
}