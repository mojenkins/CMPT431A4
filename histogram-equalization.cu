#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"
// CUDA Runtime
#include <hip/hip_runtime.h>
// Utility and system includes
#include <hip/hip_runtime_api.h>
// helper for shared that are common to CUDA Samples
#include <helper_functions.h>


__global__ void histogram_work(int img_size, int* gpu_lut, unsigned char* gpu_img_in, unsigned char* gpu_img_out){
	if (blockIdx.x + threadIdx.x < img_size){
    	gpu_img_out[blockIdx.x + threadIdx.x] = (gpu_lut[gpu_img_in[blockIdx.x + threadIdx.x]] > 255) ? 255 : gpu_lut[gpu_img_in[blockIdx.x + threadIdx.x]];
	}
}


void gpu_histogram_equalization(unsigned char * img_out, unsigned char * img_in,
                            int * hist_in, int img_size, int nbr_bin){
    int *lut = (int *)malloc(sizeof(int)*nbr_bin);
    int i, min, d, cdf;

    /* Construct the LUT by calculating the CDF */
    cdf = 0;
    min = 0;
    i = 0;
    while(min == 0){
        min = hist_in[i++];
    }
    d = img_size - min;


    for(i = 0; i < nbr_bin; i ++){
        cdf += hist_in[i];
        //lut[i] = (cdf - min)*(nbr_bin - 1)/d;
        lut[i] = (int)(((float)cdf - min)*255/d + 0.5);
        if(lut[i] < 0){
            lut[i] = 0;
        }      
    }

    // Set up pointers for gpu device memory
    unsigned char * gpu_img_in, * gpu_img_out;
    int  * gpu_lut; //, * gpu_cdf;

    // Allocate memory for img_in, hist_in, lut, and img_out
    hipMalloc( (void**)&gpu_img_in, img_size * sizeof(unsigned char) );
    hipMalloc( (void**)&gpu_img_out, img_size * sizeof(unsigned char) );
    hipMalloc( (void**)&gpu_lut, (nbr_bin) * sizeof(int) );

    // Copy img_in and cdf to gpu
    hipMemcpy( gpu_img_in, img_in, img_size * sizeof(unsigned char), hipMemcpyHostToDevice );
    hipMemcpy( gpu_lut, lut, (nbr_bin) * sizeof(int), hipMemcpyHostToDevice );
    
    // GPU version
    // call kernel
    histogram_work<<<img_size/512+1,512>>>(img_size, gpu_lut, gpu_img_in, gpu_img_out);

    // copy back (using hipMemcpy) gpu_img_out to img_out
    hipMemcpy( img_out, gpu_img_out, img_size * sizeof(unsigned char), hipMemcpyDeviceToHost );

    // free gpu memory
    hipFree(gpu_img_out);
    hipFree(gpu_img_in);
    hipFree(gpu_lut);
}

