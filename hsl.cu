#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"
// CUDA Runtime
#include <hip/hip_runtime.h>
// Utility and system includes
#include <hip/hip_runtime_api.h>
// helper for shared that are common to CUDA Samples
#include <helper_functions.h>

const int imageDepth = 256; //*** Hard coded. Only 8-bit images supported 

__global__ void rgb2hsl_kernel(int img_size, unsigned char *gpu_img_in_r, unsigned char *gpu_img_in_g, unsigned char *gpu_img_in_b, float *gpu_img_out_h, float *gpu_img_out_s, unsigned char *gpu_img_out_l) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < img_size){
        float H, S, L;
        
        // Convert RGB from [0,255] to [0,1]
        float var_r = ( (float)gpu_img_in_r[index]/(imageDepth-1) );
        float var_g = ( (float)gpu_img_in_g[index]/(imageDepth-1) );
        float var_b = ( (float)gpu_img_in_b[index]/(imageDepth-1) );
        
        // Find min and max values
        float var_min = (var_r < var_g) ? var_r : var_g;
        var_min = (var_min < var_b) ? var_min : var_b;   //min. value of RGB
        float var_max = (var_r > var_g) ? var_r : var_g;
        var_max = (var_max > var_b) ? var_max : var_b;   //max. value of RGB
        float del_max = var_max - var_min;               //Delta RGB value
        
        // Calculate L
        L = (var_max + var_min) / 2;
        
        // Calculate S
        if (del_max == 0) {
            S = 0;
        } else if (L < 0.5) {
            S = del_max/(var_max+var_min);
        } else {
            S = del_max/(2-var_max-var_min );
        }
        
        // Calculate H
        float del_r = (((var_max-var_r)/6)+(del_max/2))/del_max;
        float del_g = (((var_max-var_g)/6)+(del_max/2))/del_max;
        float del_b = (((var_max-var_b)/6)+(del_max/2))/del_max;
        
        if (del_max == 0) {
            H = 0;
        } else if (var_r == var_max) {
            H = del_b - del_g;
        } else if (var_g == var_max) {
            H = (1.0/3.0) + del_r - del_b;
        } else {
            H = (2.0/3.0) + del_g - del_r;
        }

        
        if (H < 0) {
            H += 1;
        } else if (H > 1) {
            H -= 1;
        }

        // Save HSL values to output image
        gpu_img_out_h[index] = H;
        gpu_img_out_s[index] = S;
        gpu_img_out_l[index] = (unsigned char)(L*255);
    }
}


//Convert RGB to HSL, assume R,G,B in [0, 255]
//Output H, S in [0.0, 1.0] and L in [0, 255]
HSL_IMG gpu_rgb2hsl(PPM_IMG img_in) {
    int img_size = img_in.w * img_in.h;
    HSL_IMG img_out;// = (HSL_IMG *)malloc(sizeof(HSL_IMG));
    img_out.width  = img_in.w;
    img_out.height = img_in.h;
    img_out.h = (float *)malloc(img_size * sizeof(float));
    img_out.s = (float *)malloc(img_size * sizeof(float));
    img_out.l = (unsigned char *)malloc(img_size * sizeof(unsigned char));
    
    // Set up pointers for gpu device memory
    unsigned char *gpu_img_in_r, *gpu_img_in_g, *gpu_img_in_b, *gpu_img_out_l;
    float *gpu_img_out_h, *gpu_img_out_s;
    
    // Allocate memory on GPU
    hipMalloc( (void**)&gpu_img_in_r, img_size * sizeof(unsigned char) );
    hipMalloc( (void**)&gpu_img_in_g, img_size * sizeof(unsigned char) );
    hipMalloc( (void**)&gpu_img_in_b, img_size * sizeof(unsigned char) );
    hipMalloc( (void**)&gpu_img_out_h, img_size * sizeof(unsigned char) );
    hipMalloc( (void**)&gpu_img_out_s, img_size * sizeof(unsigned char) );
    hipMalloc( (void**)&gpu_img_out_l, img_size * sizeof(unsigned char) );
    
    // Copy input image to gpu
    hipMemcpy( gpu_img_in_r, img_in.img_r, img_size * sizeof(unsigned char), hipMemcpyHostToDevice );
    hipMemcpy( gpu_img_in_g, img_in.img_g, img_size * sizeof(unsigned char), hipMemcpyHostToDevice );
    hipMemcpy( gpu_img_in_b, img_in.img_b, img_size * sizeof(unsigned char), hipMemcpyHostToDevice );
    
    // call kernel
    rgb2hsl_kernel<<<img_size/512+1,512>>>(img_size, gpu_img_in_r, gpu_img_in_g, gpu_img_in_b, gpu_img_out_h, gpu_img_out_s, gpu_img_out_l);
    
    // Copy resultant image from gpu
    hipMemcpy( img_out.h, gpu_img_out_h, img_size * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( img_out.s, gpu_img_out_s, img_size * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( img_out.l, gpu_img_out_l, img_size * sizeof(unsigned char), hipMemcpyDeviceToHost );
    
    //free gpu memory
    hipFree(gpu_img_in_r);
    hipFree(gpu_img_in_g);
    hipFree(gpu_img_in_b);
    hipFree(gpu_img_out_h);
    hipFree(gpu_img_out_s);
    hipFree(gpu_img_out_l);
    
    return img_out;
}

__global__ void hsl2rgb_kernel(int img_size, float *gpu_img_in_h, float *gpu_img_in_s, unsigned char *gpu_img_in_l, unsigned char *gpu_img_out_r, unsigned char *gpu_img_out_g, unsigned char *gpu_img_out_b) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < img_size){
				float H = gpu_img_in_h[index];
        float S = gpu_img_in_s[index];
        float L = gpu_img_in_l[index]/255.0f;
        float var1, var2;
        
        unsigned char r,g,b;
        
        if ( S == 0 )
        {
            r = L * 255;
            g = L * 255;
            b = L * 255;
        }

        else
        {
						var2 = (L < 0.5) ? (L * (1 + S)) : ((L + S) - (S * L));
						var1 = 2 * L - var2;
						
						//calculate r
						float rvH = H + (1.0f/3.0f);
						rvH = (rvH < 0) ? rvH+1 : rvH;
						rvH = (rvH > 1) ? rvH-1 : rvH;
						r = 255 * var1;
						r = ((3*rvH) < 2) ? (255 * (var1 + ( var2 - var1 ) * (( 2.0f/3.0f) - rvH) * 6)) : r;
						r = ((2*rvH) < 1) ? (255 * var2) : r;
						r = ((6*rvH) < 1) ? (255 * (var1 + ( var2 - var1 ) * 6 * rvH )) : r;

						//calculate g
						float gvH = H;
						gvH = (gvH < 0) ? gvH+1 : gvH;
						gvH = (gvH > 1) ? gvH-1 : gvH;
						g = 255 * var1;
						g = ((3*gvH) < 2) ? (255 * (var1 + ( var2 - var1 ) * (( 2.0f/3.0f) - gvH) * 6)) : g;
						g = ((2*gvH) < 1) ? (255 * var2) : g;
						g = ((6*gvH) < 1) ? (255 * (var1 + ( var2 - var1 ) * 6 * gvH )) : g;

						//calculate b
						float bvH = H - (1.0f/3.0f);
						bvH = (bvH < 0) ? bvH+1 : bvH;
						bvH = (bvH > 1) ? bvH-1 : bvH;
						b = 255 * var1;
						b = ((3*bvH) < 2) ? (255 * (var1 + ( var2 - var1 ) * (( 2.0f/3.0f) - bvH) * 6)) : b;
						b = ((2*bvH) < 1) ? (255 * var2) : b;
						b = ((6*bvH) < 1) ? (255 * (var1 + ( var2 - var1 ) * 6 * bvH )) : b;
        }

        gpu_img_out_r[index] = r;
        gpu_img_out_g[index] = g;
        gpu_img_out_b[index] = b;
		}
}

//Convert HSL to RGB, assume H, S in [0.0, 1.0] and L in [0, 255]
//Output R,G,B in [0, 255]
PPM_IMG gpu_hsl2rgb(HSL_IMG img_in) {
    PPM_IMG result;
    
		int img_size = img_in.width * img_in.height;
    result.w = img_in.width;
    result.h = img_in.height;
    result.img_r = (unsigned char *)malloc(img_size * sizeof(unsigned char));
    result.img_g = (unsigned char *)malloc(img_size* sizeof(unsigned char));
    result.img_b = (unsigned char *)malloc(img_size * sizeof(unsigned char));


    // Set up pointers for gpu device memory
    unsigned char *gpu_img_out_r, *gpu_img_out_g, *gpu_img_out_b, *gpu_img_in_l;
    float *gpu_img_in_h, *gpu_img_in_s;
    
    // Allocate memory on GPU
    hipMalloc( (void**)&gpu_img_out_r, img_size * sizeof(unsigned char) );
    hipMalloc( (void**)&gpu_img_out_g, img_size * sizeof(unsigned char) );
    hipMalloc( (void**)&gpu_img_out_b, img_size * sizeof(unsigned char) );
    hipMalloc( (void**)&gpu_img_in_h, img_size * sizeof(float) );
    hipMalloc( (void**)&gpu_img_in_s, img_size * sizeof(float) );
    hipMalloc( (void**)&gpu_img_in_l, img_size * sizeof(unsigned char) );
    
    // Copy input image to gpu
    hipMemcpy( gpu_img_in_h, img_in.h, img_size * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( gpu_img_in_s, img_in.s, img_size * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( gpu_img_in_l, img_in.l, img_size * sizeof(unsigned char), hipMemcpyHostToDevice );
    
    // call kernel
    hsl2rgb_kernel<<<img_size/512+1,512>>>(img_size, gpu_img_in_h, gpu_img_in_s, gpu_img_in_l, gpu_img_out_r, gpu_img_out_g, gpu_img_out_b);
    
    // Copy resultant image from gpu
    hipMemcpy( result.img_r, gpu_img_out_r, img_size * sizeof(unsigned char), hipMemcpyDeviceToHost );
    hipMemcpy( result.img_g, gpu_img_out_g, img_size * sizeof(unsigned char), hipMemcpyDeviceToHost );
    hipMemcpy( result.img_b, gpu_img_out_b, img_size * sizeof(unsigned char), hipMemcpyDeviceToHost );
    
    //free gpu memory
    hipFree(gpu_img_in_h);
    hipFree(gpu_img_in_s);
    hipFree(gpu_img_in_l);
    hipFree(gpu_img_out_r);
    hipFree(gpu_img_out_g);
    hipFree(gpu_img_out_b);
		
    return result;
}
