#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"
// CUDA Runtime
#include <hip/hip_runtime.h>
// Utility and system includes
#include <hip/hip_runtime_api.h>
// helper for shared that are common to CUDA Samples
#include <helper_functions.h>
#include <hip/hip_runtime.h>


__global__ void cuda_test_kernel(int* num, int *num2){
    *num += *num2;
}


bool cuda_test() {
    int *dev_num, *dev_num2; // declare device int pointers
    int host_num, host_num2, host_outputNum; // declare host integers
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // Allocate device memory
    hipMalloc( &dev_num, sizeof(int) );
    hipMalloc( &dev_num2, sizeof(int) );
    
    // Initialize host integers
    host_num = 3;
    host_num2 = 5;
    
    // Copy host integers to device
    hipMemcpy( dev_num, &host_num, sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( dev_num2, &host_num2, sizeof(int), hipMemcpyHostToDevice );
    
    // call kernel
    cuda_test_kernel<<<1,1>>>(dev_num, dev_num2);
	
    // copy back (using hipMemcpy) gpu_img_out to img_out
    hipMemcpy( &host_outputNum, dev_num, sizeof(int), hipMemcpyDeviceToHost );

    // free gpu memory
    hipFree(dev_num);
    hipFree(dev_num2);
    //printf("Processing time of test: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);
    
    int actualSum = host_num + host_num2;

    return (actualSum == host_outputNum);
}

// __global__
// void saxpy(int n, float a, float *x, float *y)
// {
//   int i = blockIdx.x*blockDim.x + threadIdx.x;
//   if (i < n) y[i] = a*x[i] + y[i];
// }
// 
// int main(void)
// {
//   int N = 1<<20;
//   float *x, *y, *d_x, *d_y;
//   x = (float*)malloc(N*sizeof(float));
//   y = (float*)malloc(N*sizeof(float));
// 
//   hipMalloc(&d_x, N*sizeof(float)); 
//   hipMalloc(&d_y, N*sizeof(float));
// 
//   for (int i = 0; i < N; i++) {
//     x[i] = 1.0f;
//     y[i] = 2.0f;
//   }
// 
//   hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
//   hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
// 
//   // Perform SAXPY on 1M elements
//   saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
// 
//   hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
// 
//   float maxError = 0.0f;
//   for (int i = 0; i < N; i++)
//     maxError = max(maxError, abs(y[i]-4.0f));
//   printf("Max error: %fn", maxError);
// }