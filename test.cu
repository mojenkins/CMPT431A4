#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"
// CUDA Runtime
#include <hip/hip_runtime.h>
// Utility and system includes
#include <hip/hip_runtime_api.h>
// helper for shared that are common to CUDA Samples
#include <helper_functions.h>
#include <hip/hip_runtime.h>


__global__ void cuda_test_kernel(int* num, int num2){
    *num +=num2;
}


void cuda_test(){
    
    int * num;
    int init_num = 5;
    int init_num2 = 4;
    int output_num = 0;
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);


    // Allocate memory for num
    hipMalloc( (void**) &num, sizeof(int) );
    
    // Copy init_num
    hipMemcpy( num, &init_num, sizeof(int), hipMemcpyHostToDevice );

    
    // call kernel
    cuda_test_kernel<<<1,1>>>(num, init_num2);

    // copy back (using hipMemcpy) gpu_img_out to img_out
    hipMemcpy( &output_num, num, sizeof(int), hipMemcpyDeviceToHost );

    // free gpu memory
    hipFree(num);
    printf("Processing time of test: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

    printf("output_num: %i \n", output_num);
}

